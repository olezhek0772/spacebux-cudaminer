#include "hip/hip_runtime.h"
/*
 * sha512 djm34
 * 
 */

/*
 * sha-512 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014  djm34
 * 
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 * 
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 * @author   phm <phm@inbox.com>
 */

#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include "uint256.h"
extern "C" {
#include "sph_sha2.h"
}

#define USE_SHARED 1
#include "cuda_helper.h"


#include "trashminer.h"

#define SPH_C64(x)    ((uint64_t)(x ## ULL))
//#define SPH_T64(x)    ((x) & SPH_C64(0xFFFFFFFFFFFFFFFF))
#define SPH_T64(x)  sph_t64(x)

__device__ __forceinline__ uint64_t SWAP64(uint64_t x)
{
	// Input:	77665544 33221100
	// Output:	00112233 44556677
	uint64_t temp[2];
	temp[0] = __byte_perm(HIWORD(x), 0, 0x0123);
	temp[1] = __byte_perm(LOWORD(x), 0, 0x0123);

	return temp[0] | (temp[1]<<32);
}

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

static __constant__ uint64_t H_512[8];

static const uint64_t H512[8] = {
	SPH_C64(0x6A09E667F3BCC908), SPH_C64(0xBB67AE8584CAA73B),
	SPH_C64(0x3C6EF372FE94F82B), SPH_C64(0xA54FF53A5F1D36F1),
	SPH_C64(0x510E527FADE682D1), SPH_C64(0x9B05688C2B3E6C1F),
	SPH_C64(0x1F83D9ABFB41BD6B), SPH_C64(0x5BE0CD19137E2179)
};
static __constant__ uint64_t K_512[80];

static const uint64_t K512[80] = {
	SPH_C64(0x428A2F98D728AE22), SPH_C64(0x7137449123EF65CD),
	SPH_C64(0xB5C0FBCFEC4D3B2F), SPH_C64(0xE9B5DBA58189DBBC),
	SPH_C64(0x3956C25BF348B538), SPH_C64(0x59F111F1B605D019),
	SPH_C64(0x923F82A4AF194F9B), SPH_C64(0xAB1C5ED5DA6D8118),
	SPH_C64(0xD807AA98A3030242), SPH_C64(0x12835B0145706FBE),
	SPH_C64(0x243185BE4EE4B28C), SPH_C64(0x550C7DC3D5FFB4E2),
	SPH_C64(0x72BE5D74F27B896F), SPH_C64(0x80DEB1FE3B1696B1),
	SPH_C64(0x9BDC06A725C71235), SPH_C64(0xC19BF174CF692694),
	SPH_C64(0xE49B69C19EF14AD2), SPH_C64(0xEFBE4786384F25E3),
	SPH_C64(0x0FC19DC68B8CD5B5), SPH_C64(0x240CA1CC77AC9C65),
	SPH_C64(0x2DE92C6F592B0275), SPH_C64(0x4A7484AA6EA6E483),
	SPH_C64(0x5CB0A9DCBD41FBD4), SPH_C64(0x76F988DA831153B5),
	SPH_C64(0x983E5152EE66DFAB), SPH_C64(0xA831C66D2DB43210),
	SPH_C64(0xB00327C898FB213F), SPH_C64(0xBF597FC7BEEF0EE4),
	SPH_C64(0xC6E00BF33DA88FC2), SPH_C64(0xD5A79147930AA725),
	SPH_C64(0x06CA6351E003826F), SPH_C64(0x142929670A0E6E70),
	SPH_C64(0x27B70A8546D22FFC), SPH_C64(0x2E1B21385C26C926),
	SPH_C64(0x4D2C6DFC5AC42AED), SPH_C64(0x53380D139D95B3DF),
	SPH_C64(0x650A73548BAF63DE), SPH_C64(0x766A0ABB3C77B2A8),
	SPH_C64(0x81C2C92E47EDAEE6), SPH_C64(0x92722C851482353B),
	SPH_C64(0xA2BFE8A14CF10364), SPH_C64(0xA81A664BBC423001),
	SPH_C64(0xC24B8B70D0F89791), SPH_C64(0xC76C51A30654BE30),
	SPH_C64(0xD192E819D6EF5218), SPH_C64(0xD69906245565A910),
	SPH_C64(0xF40E35855771202A), SPH_C64(0x106AA07032BBD1B8),
	SPH_C64(0x19A4C116B8D2D0C8), SPH_C64(0x1E376C085141AB53),
	SPH_C64(0x2748774CDF8EEB99), SPH_C64(0x34B0BCB5E19B48A8),
	SPH_C64(0x391C0CB3C5C95A63), SPH_C64(0x4ED8AA4AE3418ACB),
	SPH_C64(0x5B9CCA4F7763E373), SPH_C64(0x682E6FF3D6B2B8A3),
	SPH_C64(0x748F82EE5DEFB2FC), SPH_C64(0x78A5636F43172F60),
	SPH_C64(0x84C87814A1F0AB72), SPH_C64(0x8CC702081A6439EC),
	SPH_C64(0x90BEFFFA23631E28), SPH_C64(0xA4506CEBDE82BDE9),
	SPH_C64(0xBEF9A3F7B2C67915), SPH_C64(0xC67178F2E372532B),
	SPH_C64(0xCA273ECEEA26619C), SPH_C64(0xD186B8C721C0C207),
	SPH_C64(0xEADA7DD6CDE0EB1E), SPH_C64(0xF57D4F7FEE6ED178),
	SPH_C64(0x06F067AA72176FBA), SPH_C64(0x0A637DC5A2C898A6),
	SPH_C64(0x113F9804BEF90DAE), SPH_C64(0x1B710B35131C471B),
	SPH_C64(0x28DB77F523047D84), SPH_C64(0x32CAAB7B40C72493),
	SPH_C64(0x3C9EBE0A15C9BEBC), SPH_C64(0x431D67C49C100D4C),
	SPH_C64(0x4CC5D4BECB3E42B6), SPH_C64(0x597F299CFC657E2A),
	SPH_C64(0x5FCB6FAB3AD6FAEC), SPH_C64(0x6C44198C4A475817)
};


#define SHA3_STEP(ord,r,i) { \
	    uint64_t T1, T2; \
		int a = 8-ord; \
		T1 = SPH_T64(r[(7+a)%8] + BSG5_1(r[(4+a)%8]) + CH(r[(4+a)%8], r[(5+a)%8], r[(6+a)%8]) + K_512[i] + W[i]); \
		T2 = SPH_T64(BSG5_0(r[(0+a)%8]) + MAJ(r[(0+a)%8], r[(1+a)%8], r[(2+a)%8])); \
		r[(3+a)%8] = SPH_T64(r[(3+a)%8] + T1); \
		r[(7+a)%8] = SPH_T64(T1 + T2); \
	}

#define SHA3_STEP2(truc,ord,r,i) { \
	    uint64_t T1, T2; \
		int a = 8-ord; \
		T1 = Tone(truc,r,W,a,i); \
		T2 = SPH_T64(BSG5_0(r[(0+a)%8]) + MAJ(r[(0+a)%8], r[(1+a)%8], r[(2+a)%8])); \
		r[(3+a)%8] = SPH_T64(r[(3+a)%8] + T1); \
		r[(7+a)%8] = SPH_T64(T1 + T2); \
	}
//#define BSG5_0(x)      (ROTR64(x, 28) ^ ROTR64(x, 34) ^ ROTR64(x, 39))
#define BSG5_0(x)        xor3(ROTR64(x, 28),ROTR64(x, 34),ROTR64(x, 39))

//#define BSG5_1(x)      (ROTR64(x, 14) ^ ROTR64(x, 18) ^ ROTR64(x, 41))
#define BSG5_1(x)      xor3(ROTR64(x, 14),ROTR64(x, 18),ROTR64(x, 41))

//#define SSG5_0(x)      (ROTR64(x, 1) ^  ROTR64(x, 8) ^ SPH_T64((x) >> 7))
#define SSG5_0(x)      xor3(ROTR64(x, 1),ROTR64(x, 8),shr_t64(x,7))

//#define SSG5_1(x)      (ROTR64(x, 19) ^ ROTR64(x, 61) ^ SPH_T64((x) >> 6))
#define SSG5_1(x)      xor3(ROTR64(x, 19),ROTR64(x, 61),shr_t64(x,6))

//#define CH(X, Y, Z)    ((((Y) ^ (Z)) & (X)) ^ (Z))
#define CH(x, y, z)    xandx(x,y,z)
//#define MAJ(X, Y, Z)   (((X) & (Y)) | (((X) | (Y)) & (Z)))
#define MAJ(x, y, z)   andor(x,y,z)
static __device__ __forceinline__ uint64_t Tone(const uint64_t* sharedMemory, uint64_t r[8], uint64_t W[80], uint32_t a, uint32_t i) 
{
uint64_t h =  r[(7+a)%8];
uint64_t e=   r[(4+a)%8];
uint64_t f=   r[(5+a)%8];
uint64_t g=   r[(6+a)%8];
//uint64_t BSG51 = ROTR64(e, 14) ^ ROTR64(e, 18) ^ ROTR64(e, 41);
uint64_t BSG51 = xor3(ROTR64(e, 14),ROTR64(e, 18),ROTR64(e, 41));
//uint64_t CHl     = (((f) ^ (g)) & (e)) ^ (g);
uint64_t CHl = xandx(e,f,g);
uint64_t result = SPH_T64(h+BSG51+CHl+sharedMemory[i]+W[i]);
return result;
}


__global__ void sha512_gpu_hash_242(int threads, uint64_t startNounce, uint32_t *g_block, uint64_t *g_hash)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t *inpHash = g_block;
		
			
union {
uint8_t h1[128];
uint32_t h4[32];
uint64_t h8[16];
} hash;  

		
        
    #pragma unroll 32
	for (int i=0;i<32;i++) {
		hash.h4[i]= inpHash[i];}
		 
	
		
	uint64_t W[80]; 
        uint64_t r[8];
	uint64_t ri[8];

#pragma unroll 16
 	for (int i = 0; i < 16; i ++) {
		W[i] = SWAP64(hash.h8[i]);
	}

	W[14] = SWAP64(startNounce + thread * 0x100000000ULL);

#pragma unroll 8
	for(int i=0; i < 8; i++){
		r[i] = H_512[i];
		ri[i] = r[i];
	}
		
#pragma unroll 64
		for (int i = 16; i < 80; i ++) 
 			W[i] = SPH_T64(SSG5_1(W[i - 2]) + W[i - 7] 
				+ SSG5_0(W[i - 15]) + W[i - 16]); 

#pragma unroll 1
		for (int i = 0; i < 80; i += 8) {
#pragma unroll 8
			for (int ord=0;ord<8;ord++) {SHA3_STEP2(K_512,ord,r,i+ord);}
		}

#pragma unroll 8
		for (int i = 0; i < 8; i++) {r[i] = SPH_T64(r[i] + ri[i]);}

#if 1
#pragma unroll 32
	for (int i=0;i<32;i++) {
		hash.h4[i]= inpHash[i+32];}
		 
#pragma unroll 8
	for(int i=0; i < 8; i++){
		ri[i] = r[i];
	}

#pragma unroll 16
 	for (int i = 0; i < 16; i ++) {
		W[i] = SWAP64(hash.h8[i]);
	}
		
#pragma unroll 64
		for (int i = 16; i < 80; i ++) 
 			W[i] = SPH_T64(SSG5_1(W[i - 2]) + W[i - 7] 
				+ SSG5_0(W[i - 15]) + W[i - 16]); 

#pragma unroll 1
		for (int i = 0; i < 80; i += 8) {
#pragma unroll 8
			for (int ord=0;ord<8;ord++) {SHA3_STEP2(K_512,ord,r,i+ord);}
		}

#pragma unroll 8
	for (int i = 0; i < 8; i++) {r[i] = SPH_T64(r[i] + ri[i]);}

#endif

#pragma unroll 8
	for(int i=0;i<8;i++) {	
		hash.h8[i] = SWAP64(r[i]);}

      
      #pragma unroll 16
      for (int u = 0; u < 8; u ++) 
            g_hash[u*threads+thread] = hash.h8[u];    
 }
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void sha512_cpu_init(int thr_id, int threads, ctx* pctx)
{

    hipMemcpyToSymbol(HIP_SYMBOL(K_512),K512,80*sizeof(uint64_t),0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(H_512),H512,sizeof(H512),0, hipMemcpyHostToDevice);
	
    gpuErrchk(hipMalloc( (void**)&pctx->sha512_dblock, 256 )); 
}


__host__ void sha512_cpu_hash_242(int thr_id, int threads, uint64_t startNounce, uint32_t* dblock, uint64_t *d_hash)
{

	const int threadsperblock = 512; // Alignment mit mixtab Gr\F6sse. NICHT \C4NDERN

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
//	dim3 grid(1);
//	dim3 block(1);
//	size_t shared_size = 80*sizeof(uint64_t);
	size_t shared_size =0;
	sha512_gpu_hash_242<<<grid, block, shared_size>>>(threads, startNounce, dblock, d_hash);

      //  hipStreamSynchronize(0);
	MyStreamSynchronize(NULL, 2, thr_id);
}

void sha512_scanhash(int throughput, uint64_t startNounce, CBlockHeader *hdr, uint64_t *d_hash, ctx* pctx){
	char block[256];
	uint64_t hash[8];

	memset(block,0,sizeof(block));
	memcpy(block,hdr,sizeof(*hdr));

	block[122] = 0x80;
	((uint64_t*)block)[256/8 - 1] = swap_uint64(976);

	gpuErrchk(hipMemcpyAsync( pctx->sha512_dblock, block, sizeof(block), hipMemcpyHostToDevice, 0 )); 

	sha512_cpu_hash_242(pctx->thr_id,throughput,startNounce,pctx->sha512_dblock,d_hash);

}


